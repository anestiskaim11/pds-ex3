
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

int *unroll(int **ising, int n){
    int *ising1d = (int *)malloc(n * n * sizeof(int));
    for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            ising1d[i*n + j] = ising[i][j];
        }
    }

    return ising1d;
}
void swap(int  **a, int  **b) {
  int  *tmp = *a;
  *a = *b;
  *b = tmp;
}

__global__ void moment(int *ising, int *newising, int n){
    //printf("%d\n", blockIdx.x);
    //if(threadIdx.x == 1023) printf("%d\n", threadIdx.x);
    int sum = ising[(blockIdx.x*1024 + threadIdx.x+n)%(n*n)] + ising[(blockIdx.x*1024 + threadIdx.x-n)%(n*n)]
    + ising[blockIdx.x*1024 + threadIdx.x + 1 - n*(threadIdx.x%n == n - 1)]
    + ising[blockIdx.x*1024 + threadIdx.x - 1 + n*(threadIdx.x%n == 0)]
    + ising[blockIdx.x*1024 + threadIdx.x];
    if(sum > 0)
        newising[blockIdx.x*1024 + threadIdx.x] = 1 ;
    else
        newising[blockIdx.x*1024 + threadIdx.x] = -1 ;
    
}

int main(int argc, char **argv){

    //size of Ising model
    int n = 2048 ;
    // number of iterations
    int k = 100 ;

    srand(time(NULL));

    int *ising = (int *) malloc(n * n * sizeof(int));
    for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j ++){
            ising[i*n + j] = rand() % 2 ;
            if(ising[i*n + j] == 0){
                ising[i*n + j] = -1;
            }
        }
    }

    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , ising[i*n + j]);
        }
        printf("\n");
    }
    printf("\n");*/

    int *newising = (int *)malloc(n * n * sizeof(int));
    
    int *d_ising;
    int *d_newising;
    int size = n * n * sizeof(int);
    
    //allocate on gpu
    hipMalloc((void**)&d_ising, size);
    hipMalloc((void**)&d_newising, size);
    
    
    struct timeval start, end;
    double time;
    

    int blocks = (n*n - 1)/1024 + 1;
    for(int l = 0 ; l < k ; l++){
        //copy data to gpu
        hipMemcpy(d_ising, ising, size, hipMemcpyHostToDevice);
        //call function on gpu with n*n threads
        gettimeofday(&start, NULL);
        moment<<<blocks,n*n/blocks>>>(d_ising, d_newising, n);
        gettimeofday(&end, NULL);
        //copy result from gpu
        hipMemcpy(newising, d_newising, size, hipMemcpyDeviceToHost);
        time += (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec);

        swap(&ising,&newising);
        
    }

    /*for(int i = 0 ; i < n ; i++){
            for(int j = 0 ; j < n ; j++){
                printf("%d " , ising[i*n + j]);
            }
            printf("\n");
        }
        printf("\n");*/

    printf("time: %f\n", time);
    //free pointers
    free(ising);
    free(newising);
    hipFree(d_ising);
    hipFree(d_newising);
    return 0 ;
}
